#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <cmath>
#include <corecrt_malloc.h>
#include "gpuKernel.cuh"

__global__ void transposeMat(int* a, int* b ,unsigned int size)
{
    int i = blockIdx.x * size + threadIdx.x;
    int j = threadIdx.x * size + blockIdx.x;

    b[j] = a[i];
}

__global__ void matFromProAtt(int* mat , int* res, double* pro, double* att, int size)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    res[i * size + j] = lround((double)mat[i * size + j] * pro[i] * att[j]);

//    printf("I am %d and my result id %d\n", i * size + j, res[i * size + j]);
}

int* launchGPUTranspose(int* a, unsigned int size)
{
    int* dev_a = 0;
    int* dev_res = 0;
    int* res = a;

    hipError_t status;

    status = hipSetDevice(0);
    if (status != hipSuccess)
    {
        fprintf(stderr,"Can't Find GPU");
        return NULL;
    }

    status = hipMalloc((void**)&dev_a,size * size *sizeof(int));
    status = hipMalloc((void**)&dev_res, size * size * sizeof(int));

    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory");
        goto FreeData;
    }

    status = hipMemcpy(dev_a,a,size * size * sizeof(int),hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory");
        goto FreeData;
    }

    status = hipMemcpy(dev_a,a,sizeof(int) * size * size, hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory");
        goto FreeData;
    }

    transposeMat << <size, size >> > (dev_a,dev_res,size);

    status = hipGetLastError();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Kernel didn't launch correctly");
        goto FreeData;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Error happened in a block");
        goto FreeData;
    }

    status = hipMemcpy(res, dev_res, size * size * sizeof(int), hipMemcpyDeviceToHost);

    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory From GPU");
        goto FreeData;
    }

FreeData:
    
    hipFree(dev_res);
    hipFree(dev_a);

    status = hipDeviceReset();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Reset Cuda");
    }

    return res;
}

int* launchCalculateMatFromProAtt(int* mat, double* pro, double* att,unsigned int size)
{
    int* res = mat;
    int* dev_mat = 0, * dev_res = 0;
    double* dev_pro = 0, * dev_att = 0;

    hipError_t status;

    status = hipSetDevice(0);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Find GPU");
        goto FreeData;
    }

    status = hipMalloc((void**)&dev_mat, size * size * sizeof(int));
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory on GPU");
        goto FreeData;
    }

    status = hipMalloc((void**)&dev_res, size * size * sizeof(int));
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory on GPU");
        goto FreeData;
    }

    status = hipMalloc((void**)&dev_pro, size * sizeof(double));
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory on GPU");
        goto FreeData;
    }

    status = hipMalloc((void**)&dev_att, size * sizeof(double));
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Allocate Memory on GPU");
        goto FreeData;
    }

    status = hipMemcpy(dev_mat, mat, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory");
        goto FreeData;
    }

    status = hipMemcpy(dev_pro, pro, size * sizeof(double), hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory");
        goto FreeData;
    }

    status = hipMemcpy(dev_att, pro, size * sizeof(double), hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory");
        goto FreeData;
    }

    matFromProAtt << <size, size >> > (dev_mat, dev_res, dev_pro, dev_att, size);

    status = hipGetLastError();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Launch Kernel");
        goto FreeData;
    }

    status = hipDeviceSynchronize();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Error with a block");
        goto FreeData;
    }

    status = hipMemcpy(res,dev_res,size * size * sizeof(int),hipMemcpyDeviceToHost);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Copy Memory From GPU");
        goto FreeData;
    }

FreeData:

    hipFree(dev_att);
    hipFree(dev_mat);
    hipFree(dev_pro);
    hipFree(dev_res);

    status = hipDeviceReset();
    if (status != hipSuccess)
    {
        fprintf(stderr, "Can't Reset Cuda");
    }

    return res;
}

